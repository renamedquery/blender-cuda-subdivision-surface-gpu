// trying to get the catmull clark subdiv method working in c++ so I can translate it to cuda


#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>

using namespace std;

// from https://forums.developer.nvidia.com/t/throughput-test-add-mul-mod-giving-strange-result/32021
// remove when done
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

struct vec3 {
    double x = 0;
    double y = 0;
    double z = 0;
    bool modified = false;
    int status = 0;
};

struct vec2 {
    double x = 0;
    double y = 0;
};

struct vertex {
    vec3 position;
    vec2 textureCoordinate;
    vec3 normal;
    int id;
    int neighboringFaceIDs[4];
    bool alreadyAveraged = false;
};

struct quadFace {
    int vertexIndex[4];
    int edgeVertexIndex[4];
    int textureIndex[4];
    int normalIndex[4];
    vec3 midpoint;
    int midpointVertID;
    int edgeSimplificationMatches = 0;
};

__device__ int threadID;

__device__ vertex *objVertices;
__device__ quadFace *objFaces;
__device__ vec3 *faceMidpoints;
__device__ quadFace *newFaces;
__device__ vertex *newVertices; 

std::vector<std::string> stringSplit(std::string string, char delimiter) {

    std::vector<std::string> splitString;
    std::string currentString = "";

    for (int i = 0; i < string.length(); i++) {
        if (string[i] == delimiter) {

            splitString.push_back(currentString);
            currentString = "";
        } else {

            currentString += string[i];

            if (i + 1 == string.length()) {
                splitString.push_back(currentString);
            }
        }
    }

    return splitString;
}

// currently only reads verts, faces and edges are todo
void readObj(std::string path, std::vector<vertex>& vertices, std::vector<quadFace>& faces) {
    
    std::ifstream objFile(path);

    // tell the program to not count new lines
    objFile.unsetf(std::ios_base::skipws);

    std::string objFileLine;

    int dataCount_v = 0;
    int id = 0;

    while (getline(objFile, objFileLine)) {

        std::stringstream ss{objFileLine};
        char objFileLineChar;
        ss >> objFileLineChar;

        std::vector<std::string> lineDataSplitBySpaces = stringSplit(objFileLine, ' ');
        std::string lineType = lineDataSplitBySpaces[0];

        vertex currentVert;

        bool wasVert = false;
        int vertType = 0; // 0 = none, 1 = vert, 2 = texture coordinate, 3 = normal vert

        if (lineType.compare("v") == 0) {
            currentVert.position.x = std::stod(lineDataSplitBySpaces[1]);
            currentVert.position.y = std::stod(lineDataSplitBySpaces[2]);
            currentVert.position.z = std::stod(lineDataSplitBySpaces[3]);
            currentVert.position.modified = true;
            currentVert.id = id;

            wasVert = true;
            vertType = 1;
            dataCount_v++;

        } else if (lineType.compare("f") == 0) {

            quadFace currentFace;

            for (int i = 1; i < lineDataSplitBySpaces.size(); i++) {
                
                std::vector<std::string> lineDataSplitBySlashes = stringSplit(lineDataSplitBySpaces[i], '/');

                // vertex_index, texture_index, normal_index
                currentFace.vertexIndex[i - 1] = std::stod(lineDataSplitBySlashes[0]) - 1;
                currentFace.textureIndex[i - 1] = 0;
                currentFace.normalIndex[i - 1] = 0;

            }

            faces.push_back(currentFace);
        }

        if (wasVert) {

            if (currentVert.id < dataCount_v) vertices.push_back(currentVert);

            // check for which part of the vert has already been written to since the verts are written before the normals verts
            // if the vert type is 1 (v) and the vert hasnt been modified on the verts array
            if (vertType == 1 && !vertices[(dataCount_v - 1)].position.modified) {

                vertices[(dataCount_v - 1)].position.x = currentVert.position.x;
                vertices[(dataCount_v - 1)].position.y = currentVert.position.y;
                vertices[(dataCount_v - 1)].position.z = currentVert.position.z;
                vertices[(dataCount_v - 1)].position.modified = true;
            }

            id++;
        }
    }

    objFile.close();
}

// can not be gpu accelerated - is sequental
void writeObj(std::string path, std::vector<vertex> vertices, std::vector<quadFace> faces) {

    std::ofstream objFile;
    objFile.open(path, ios::out | ios::trunc);

    objFile << "o EXPERIMENTAL_MESH" << endl;

    for (int i = 0; i < vertices.size(); i++) {
        
        objFile << "v " << std::to_string(vertices[i].position.x) << " " << std::to_string(vertices[i].position.y) << " " << std::to_string(vertices[i].position.z) << endl;
    }

    for (int i = 0; i < faces.size(); i++) {

        objFile << "f ";

        for (int j = 0; j < 4; j++) {

            objFile << std::to_string(faces[i].vertexIndex[j] + 1) << " ";
        }

        objFile << endl;
    }

    objFile.close();
}

__global__ void catmullClarkFacePointsAndEdges(int facesSize_lcl, int maxVertsAtStart_lcl, int totalMaxVerts_lcl) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i > facesSize_lcl) return;

    quadFace currentSubdividedFaces[4];
    
    for (int j = 0; j < 4; j++) currentSubdividedFaces[j].vertexIndex[3] = *&objFaces[i].midpointVertID; // face point [0] will be the center of the subdivided face

    // vertex ids for the edges

    int vertexIDs[4];

    for (int j = 0; j < 4; j++) {

        vec3 edgeAveragePoint;

        vertex edgePoint;

        edgeAveragePoint.x = (*&objVertices[*&objFaces[i].vertexIndex[(j + 1) % 4]].position.x + *&objVertices[*&objFaces[i].vertexIndex[(j + 0) % 4]].position.x) / 2;
        edgeAveragePoint.y = (*&objVertices[*&objFaces[i].vertexIndex[(j + 1) % 4]].position.y + *&objVertices[*&objFaces[i].vertexIndex[(j + 0) % 4]].position.y) / 2;
        edgeAveragePoint.z = (*&objVertices[*&objFaces[i].vertexIndex[(j + 1) % 4]].position.z + *&objVertices[*&objFaces[i].vertexIndex[(j + 0) % 4]].position.z) / 2;

        printf("%d\n", edgeAveragePoint.x);

        currentSubdividedFaces[j].vertexIndex[1] = *&objFaces[i].vertexIndex[(j + 0) % 4];

        // find the averages for the face points

        edgePoint.id = maxVertsAtStart_lcl + (i * 5) + (j + 1);

        vertexIDs[j] = edgePoint.id;

        currentSubdividedFaces[j].vertexIndex[0] = edgePoint.id;
        currentSubdividedFaces[(j + 1) % 4].vertexIndex[2] = edgePoint.id;

        objVertices[vertexIDs[j]].position = edgeAveragePoint;
        objFaces[i].edgeVertexIndex[j] = vertexIDs[j];
    }

    for (int j = 0; j < 4; j++) {

        newFaces[(i * 4) + j] = currentSubdividedFaces[j];
    }

    objVertices[*&objFaces[i].midpointVertID].position = faceMidpoints[i];
}
/*
__global__
void averageCornerVertices(int facesSize, const int * const & i) {

    for (int j = 0; j < 4; j++) {

        int matchedPoints = 0;
        int neighboringFaceIDs[4];

        vec3 neighboringFaceMidpointsAverage;
        vec3 edgeMidpointsAverage;
        vec3 finalMidpointAverage;

        for (int k = 0; k < facesSize; k++) {

            for (int l = 0; l < 4; l++) {

                if (objFaces[i].vertexIndex[j] == objFaces[k].vertexIndex[l]) {

                    neighboringFaceIDs[matchedPoints] = k;

                    edgeMidpointsAverage.x += (objVertices[objFaces[i].vertexIndex[j]].position.x + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.x) / 2;
                    edgeMidpointsAverage.y += (objVertices[objFaces[i].vertexIndex[j]].position.y + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.y) / 2;
                    edgeMidpointsAverage.z += (objVertices[objFaces[i].vertexIndex[j]].position.z + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.z) / 2;

                    matchedPoints++;
                }
            }
        }

        for (int k = 0; k < 4; k++) {

            neighboringFaceMidpointsAverage.x += faceMidpoints[neighboringFaceIDs[k]].x;
            neighboringFaceMidpointsAverage.y += faceMidpoints[neighboringFaceIDs[k]].y;
            neighboringFaceMidpointsAverage.z += faceMidpoints[neighboringFaceIDs[k]].z;
        }

        neighboringFaceMidpointsAverage.x /= matchedPoints;
        neighboringFaceMidpointsAverage.y /= matchedPoints;
        neighboringFaceMidpointsAverage.z /= matchedPoints;

        edgeMidpointsAverage.x /= matchedPoints;
        edgeMidpointsAverage.y /= matchedPoints;
        edgeMidpointsAverage.z /= matchedPoints;

        finalMidpointAverage.x = (neighboringFaceMidpointsAverage.x + edgeMidpointsAverage.x) / 2;
        finalMidpointAverage.y = (neighboringFaceMidpointsAverage.y + edgeMidpointsAverage.y) / 2;
        finalMidpointAverage.z = (neighboringFaceMidpointsAverage.z + edgeMidpointsAverage.z) / 2;

        newVertices[objFaces[i].vertexIndex[j]].position = edgeMidpointsAverage; // find a way to get the finalMidpointAverage to work properly
    }
}
*/
/*
__global__
void mergeByDistance(std::vector<vertex>& vertices, int i, int& completeThreads, std::vector<quadFace>& faces) {

    if (faces[i].edgeSimplificationMatches < 4) {

        for (int j = 0; j < faces.size(); j++) {

            if (!(faces[j].edgeSimplificationMatches < 4)) continue;

            int matches = 0;

            for (int k = 0; k < 4; k ++) {

                if (!(faces[j].edgeSimplificationMatches < 4)) continue;

                for (int l = 0; l < 4; l++) {

                    if (!(faces[j].edgeSimplificationMatches < 4)) continue;

                    if (
                        vertices[faces[i].vertexIndex[k]].position.x == vertices[faces[j].vertexIndex[l]].position.x &&
                        vertices[faces[i].vertexIndex[k]].position.y == vertices[faces[j].vertexIndex[l]].position.y &&
                        vertices[faces[i].vertexIndex[k]].position.z == vertices[faces[j].vertexIndex[l]].position.z &&
                        faces[i].vertexIndex[k] != faces[j].vertexIndex[l]
                    ) {
                        
                        matches++;

                        threadingMutex.lock();
                        faces[j].edgeSimplificationMatches++;
                        faces[i].edgeSimplificationMatches++;
                        threadingMutex.unlock();

                        if (!(matches < 1) && vertices[faces[j].vertexIndex[l]].position.status == 0) {

                            threadingMutex.lock();
                            vertices[faces[i].vertexIndex[k]].position.status = 1;
                            faces[j].vertexIndex[l] = faces[i].vertexIndex[k];
                            threadingMutex.unlock();
                        }
                    }
                }
            }
        }
    }

    threadingMutex.lock();
    completeThreads++;
    threadingMutex.unlock();
}
*/

int main (void) {
    
    std::string objPath = "./testMesh.obj";
    std::string objOutputPath = "./testMeshOutput.obj";

    std::vector<vertex> vertices;
    std::vector<quadFace> faces;

    const int blockSize = 256;

    readObj(objPath, vertices, faces);

    int facesSize_host = faces.size();
    int verticesSize_host = vertices.size();
    int totalNewVertsToAllocate_host = facesSize_host * 5;

    vertex *objVertices_host = new vertex[verticesSize_host + totalNewVertsToAllocate_host]; 
    quadFace *objFaces_host = new quadFace[facesSize_host]; 
    vec3 *faceMidpoints_host = new vec3[facesSize_host]; 
    quadFace *newFaces_host = new quadFace[facesSize_host * 4]; 
    vertex *newVertices_host = new vertex[verticesSize_host + (facesSize_host * 5)]; 

    for (int j = 0; j < verticesSize_host; j++) {

        objVertices_host[j] = vertices[j];
    } 

    for (int j = verticesSize_host; j < verticesSize_host + totalNewVertsToAllocate_host; j++) {

        vertex tmp;
        objVertices_host[j] = tmp;
    }

    for (int j = 0; j < facesSize_host; j++) {

        objFaces_host[j] = faces[j];
    }

    for (int j = 0; j < facesSize_host; j++) {

        vec3 faceAverageMiddlePoint;

        faceAverageMiddlePoint.x = (
            (objVertices_host[objFaces_host[j].vertexIndex[0]].position.x) + 
            (objVertices_host[objFaces_host[j].vertexIndex[1]].position.x) + 
            (objVertices_host[objFaces_host[j].vertexIndex[2]].position.x) + 
            (objVertices_host[objFaces_host[j].vertexIndex[3]].position.x)
        ) / 4;

        faceAverageMiddlePoint.y = (
            (objVertices_host[objFaces_host[j].vertexIndex[0]].position.y) + 
            (objVertices_host[objFaces_host[j].vertexIndex[1]].position.y) + 
            (objVertices_host[objFaces_host[j].vertexIndex[2]].position.y) + 
            (objVertices_host[objFaces_host[j].vertexIndex[3]].position.y)
        ) / 4;

        faceAverageMiddlePoint.z = (
            (objVertices_host[objFaces_host[j].vertexIndex[0]].position.z) + 
            (objVertices_host[objFaces_host[j].vertexIndex[1]].position.z) + 
            (objVertices_host[objFaces_host[j].vertexIndex[2]].position.z) + 
            (objVertices_host[objFaces_host[j].vertexIndex[3]].position.z)
        ) / 4;

        faceMidpoints_host[j] = faceAverageMiddlePoint;
        objFaces_host[j].midpointVertID = verticesSize_host + (j * 5);
    }

    CUDA_CHECK_RETURN(hipMalloc((void**)&objVertices_host, sizeof(objVertices_host)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&objFaces_host, sizeof(objFaces_host)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&faceMidpoints_host, sizeof(faceMidpoints_host)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&newFaces_host, sizeof(newFaces_host)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&newVertices_host, sizeof(newVertices_host)));

    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(objVertices), &objVertices_host, sizeof(objVertices_host)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(objFaces), &objFaces_host, sizeof(objFaces_host)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(faceMidpoints), &faceMidpoints_host, sizeof(faceMidpoints_host)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(newFaces), &newFaces_host, sizeof(newFaces_host)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(newVertices), &newVertices_host, sizeof(newVertices_host)));

    catmullClarkFacePointsAndEdges<<<(facesSize_host + blockSize - 1) / blockSize, blockSize>>>(facesSize_host, verticesSize_host, verticesSize_host + totalNewVertsToAllocate_host);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    //facesSize_host *= 4;

    //averageCornerVertices<<<(facesSize + blockSize - 1) / blockSize, blockSize>>>(facesSize, *&threadID);

    //std::thread(averageCornerVertices, std::ref(vertices), std::ref(newVertices), std::ref(faces), i, std::ref(completeThreads), maxVertsAtStart, std::ref(faceMidpoints), std::ref(localFaceMidpointVertIDs)).detach();

    /*
    //catmullClarkSubdiv(objVertices, objFaces, MAX_CORES, objFaces.size());
    const int originalMaxVertID = maxVertsAtStart; // for finding the original non-interpolated verts

    // face points and edge points

    int completeThreads = 0;
    int threadCountOverrunHalts = 0; // the amount of times the program has to stop spawning new threads to wait for the old ones to fall below the MAX_CORES limit

    
    std::atomic<int> workInProgressThreads(0);

    // each thread adds 5 new face points
    // calculate the total new points

    int totalNewVertsToAllocate = faces.size() * 5;

    std::vector<quadFace> newFaces;

    // make new placeholder vertices

    for (int i = 0; i < totalNewVertsToAllocate; i++) {
        
        vertex vert;
        vertices.push_back(vert);
    }

    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] SPAWNING " << faces.size() << " THREADS" << endl;

    std::vector<vec3> faceMidpoints;
    std::vector<int> localFaceMidpointVertIDs;

    for (int j = 0; j < faces.size(); j++) {

        vec3 faceAverageMiddlePoint;

        faceAverageMiddlePoint.x = (
            (vertices[faces[j].vertexIndex[0]].position.x) + 
            (vertices[faces[j].vertexIndex[1]].position.x) + 
            (vertices[faces[j].vertexIndex[2]].position.x) + 
            (vertices[faces[j].vertexIndex[3]].position.x)
        ) / 4;

        faceAverageMiddlePoint.y = (
            (vertices[faces[j].vertexIndex[0]].position.y) + 
            (vertices[faces[j].vertexIndex[1]].position.y) + 
            (vertices[faces[j].vertexIndex[2]].position.y) + 
            (vertices[faces[j].vertexIndex[3]].position.y)
        ) / 4;

        faceAverageMiddlePoint.z = (
            (vertices[faces[j].vertexIndex[0]].position.z) + 
            (vertices[faces[j].vertexIndex[1]].position.z) + 
            (vertices[faces[j].vertexIndex[2]].position.z) + 
            (vertices[faces[j].vertexIndex[3]].position.z)
        ) / 4;

        faceMidpoints.push_back(faceAverageMiddlePoint);
        localFaceMidpointVertIDs.push_back(maxVertsAtStart + (j * 5) + 0);
    }

    for (int i = 0; i < faces.size(); i++) {

        workInProgressThreads++;
        std::thread(catmullClarkFacePointsAndEdges, std::ref(vertices), std::ref(faces), std::ref(newFaces), originalMaxVertID, i, std::ref(completeThreads), maxVertsAtStart, std::ref(faceMidpoints), std::ref(localFaceMidpointVertIDs)).detach();

        if (i % 100 == 0) {

            std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] " << std::to_string(((float)i / (float)faces.size()) * 100) << "% DONE" << endl;
        }

        while (workInProgressThreads - completeThreads > MAX_CORES) {
            
            threadCountOverrunHalts++;

            if (workInProgressThreads - completeThreads <= MAX_CORES) break;
        }
    };

    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] THREAD SPAWNING IS DONE" << endl;
    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] threadCountOverrunHalts=" << std::to_string(threadCountOverrunHalts) << endl;
    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] WAITING FOR THREADS TO FINISH" << endl;

    while (true) {

        if (workInProgressThreads <= completeThreads) break;
    }

    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] ALL THREADS ARE DONE" << endl;

    std::cout << "[CPU] [averageCornerVertices()] SPAWNING " << originalMaxVertID << " THREADS" << endl;

    completeThreads = 0;
    workInProgressThreads = 0;
    threadCountOverrunHalts = 0;

    auto newVertices = vertices;

    // neighboring face midpoint gathering
    for (int i = 0; i < faces.size(); i++) {

        workInProgressThreads++;
        std::thread(averageCornerVertices, std::ref(vertices), std::ref(newVertices), std::ref(faces), i, std::ref(completeThreads), maxVertsAtStart, std::ref(faceMidpoints), std::ref(localFaceMidpointVertIDs)).detach();

        //if (i % 100 == 0) {

            //std::cout << "[CPU] [averageCornerVertices()] " << std::to_string(((float)i / (float)faces.size()) * 100) << "% DONE" << endl;
        //}

        while (workInProgressThreads - completeThreads > MAX_CORES) {
            
            threadCountOverrunHalts++;

            if (workInProgressThreads - completeThreads <= MAX_CORES) break;
        }
    }

    //std::cout << "[CPU] [averageCornerVertices()] THREAD SPAWNING IS DONE" << endl;
    //std::cout << "[CPU] [averageCornerVertices()] threadCountOverrunHalts=" << std::to_string(threadCountOverrunHalts) << endl;
    //std::cout << "[CPU] [averageCornerVertices()] WAITING FOR THREADS TO FINISH" << endl;

    while (true) {

        if (workInProgressThreads <= completeThreads) break;
    }

    vertices = newVertices;

    //std::cout << "[CPU] [averageCornerVertices()] ALL THREADS ARE DONE" << endl;

    //std::cout << "[CPU] [mergeByDistance()] SPAWNING " << originalMaxVertID << " THREADS" << endl;

    completeThreads = 0;
    workInProgressThreads = 0;
    threadCountOverrunHalts = 0;

    faces.clear();
    faces = newFaces;

    // neighboring face midpoint gathering
    for (int i = 0; i < faces.size(); i++) {

        workInProgressThreads++;
        //std::thread(mergeByDistance, std::ref(vertices), i, std::ref(completeThreads), std::ref(faces)).detach();

        //if (i % (100 * 4) == 0) {

            //std::cout << "[CPU] [mergeByDistance()] " << std::to_string(((float)i / (float)faces.size()) * 100) << "% DONE" << endl;
        //}

        while (workInProgressThreads - completeThreads > MAX_CORES) {
            
            threadCountOverrunHalts++;

            if (workInProgressThreads - completeThreads <= MAX_CORES) break;
        }
    }

    //std::cout << "[CPU] [mergeByDistance()] THREAD SPAWNING IS DONE" << endl;
    //std::cout << "[CPU] [mergeByDistance()] threadCountOverrunHalts=" << std::to_string(threadCountOverrunHalts) << endl;
    //std::cout << "[CPU] [mergeByDistance()] WAITING FOR THREADS TO FINISH" << endl;

    while (true) {

        if (workInProgressThreads <= completeThreads) break;
    }

    //std::cout << "[CPU] [mergeByDistance()] ALL THREADS ARE DONE" << endl;

    vertCount = std::to_string(objVertices.size());
    faceCount = std::to_string(objFaces.size());

    writeObj(objOutputPath, objVertices, objFaces);

    //printVerts(objVertices);
    //printFaces(objFaces, objVertices);*/

    return 0;
}